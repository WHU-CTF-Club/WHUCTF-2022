#include <hip/hip_runtime.h>


#include <cstdio>
#include <cstdlib>  // exit
#include <cstring>  // memcmp
#include <cstdint>  // uint32_t
#include <string>   // std::string
#include <iostream> // std::cout

#define NV_FAILED(status) (status != hipSuccess)
#define NV_SHOWERR(status) std::cout << "Possible reason: " << hipGetErrorString(status) << std::endl

#ifdef _MSC_VER
#define NV_INLINE __forceinline
#else
#define NV_INLINE inline
#endif

__global__ void encrypt(uint32_t* v, uint32_t* k) 
{
    v += threadIdx.x * 2;
    uint32_t v0 = v[0], v1 = v[1], sum = 0, i;
    uint32_t delta = 0x9e3779b9;
    uint32_t k0 = k[0], k1 = k[1], k2 = k[2], k3 = k[3];
    for (i = 0; i < 18; i++) {
        sum += delta;
        v0 += ((v1 << 4) + k0) ^ (v1 + sum) ^ ((v1 >> 5) + k1);
        v1 += ((v0 << 4) + k2) ^ (v0 + sum) ^ ((v0 >> 5) + k3);
    }
    v[0] = v0; v[1] = v1;
}

uint32_t* g_v = nullptr;
uint32_t* g_k = nullptr;

NV_INLINE void deallocMemory()
{
    if (g_k)
        hipFree(g_k);
    if (g_v)
        hipFree(g_v);
}
NV_INLINE void myExit(int exitCode)
{
    deallocMemory();
    exit(exitCode);
}
NV_INLINE void initDevice()
{
    hipError_t status;
    
    int cudaDeviceCount;
    status = hipGetDeviceCount(&cudaDeviceCount);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to get GPU count!" << std::endl;
        NV_SHOWERR(status);
        myExit(1);
    }

    if (cudaDeviceCount < 1)
    {
        std::cout << "Failed to find an available GPU! This program requires at least one GPU support CUDA!" << std::endl;
        NV_SHOWERR(status);
        myExit(1);
    }

    const int deviceId = cudaDeviceCount - 1;

    status = hipSetDevice(deviceId);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to select a GPU!" << std::endl;
        NV_SHOWERR(status);
        myExit(1);
    }

    hipDeviceProp_t prop;
    status = hipGetDeviceProperties(&prop, deviceId);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to get GPU property!" << std::endl;
        NV_SHOWERR(status);
        myExit(1);
    }
    std::cout << "Successfully select GPU: " << prop.name << std::endl;
}
NV_INLINE bool checkFormat(std::string& s)
{
    if (s.length() != 32)
        return false;

    if (0 != memcmp("MAYCTF{", s.c_str(), 7))
        return false;

    if (s.back() != '}')
        return false;

    return true;
}
NV_INLINE void initMemory(const char* flag)
{
    hipError_t status;

    status = hipMalloc(&g_v, 32);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to malloc memory for g_v!" << std::endl;
        NV_SHOWERR(status);
        myExit(2);
    }
    
    status = hipMalloc(&g_k, 16);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to malloc memory for g_k!" << std::endl;
        NV_SHOWERR(status);
        myExit(2);
    }

    status = hipMemcpy(g_v, flag, 32, hipMemcpyHostToDevice);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to copy data to g_v!" << std::endl;
        NV_SHOWERR(status);
        myExit(2);
    }

    constexpr char key[] = "MAYCTF_REV_CUDA!";
    status = hipMemcpy(g_k, key, 16, hipMemcpyHostToDevice);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to copy data to g_k!" << std::endl;
        NV_SHOWERR(status);
        myExit(2);
    }
}
NV_INLINE void executeEncrypt(unsigned char sn[])
{
    hipError_t status;

    encrypt<<<1, 4>>>(g_v, g_k);
    status = hipGetLastError();
    if (NV_FAILED(status))
    {
        std::cout << "Failed to verify data!" << std::endl;
        NV_SHOWERR(status);
        myExit(3);
    }

    status = hipDeviceSynchronize();
    if (NV_FAILED(status))
    {
        std::cout << "Failed to call cudaDeviceSynchronize!" << std::endl;
        NV_SHOWERR(status);
        myExit(3);
    }
    
    status = hipMemcpy(sn, g_v, 32, hipMemcpyDeviceToHost);
    if (NV_FAILED(status))
    {
        std::cout << "Failed to copy data back to host!" << std::endl;
        NV_SHOWERR(status);
        myExit(3);
    }
}
NV_INLINE bool checkAnswer(unsigned char sn[])
{
    unsigned char ans[32] =
    {
        0xef, 0x6e, 0xec, 0x60, 0xeb, 0x84, 0x92, 0x00,
        0x42, 0x2f, 0x1a, 0x22, 0x20, 0xb2, 0xb4, 0x83,
        0xa1, 0xe8, 0x61, 0x06, 0xb3, 0x76, 0xaf, 0x8d,
        0xc4, 0x8f, 0x55, 0x8c, 0xc8, 0x71, 0x9b, 0x86,
    };

    return 0 == memcmp(sn, ans, 32);
}

// modified TEA-Encrypt which only has 18 rounds
// key = MAYCTF_REV_CUDA!
// flag = MAYCTF{NviDiA_CuDA_GPU_ProGrAm!}
int main()
{
    std::cout << "Initializing device..." << std::endl;
    initDevice();
    
    std::cout << "Please input your flag:" << std::endl;
    std::string flag;
    std::cin >> flag;
    if (!checkFormat(flag))
    {
        std::cout << "You don't understand Reverse of course!" << std::endl;
        myExit(114514);
    }

    std::cout << "Hold on! I'm verifying your flag...";
    initMemory(flag.c_str());

    unsigned char sn[32] = { 0 };
    executeEncrypt(sn);
    std::cout << " Done.\n" << std::endl;

    if (checkAnswer(sn))
        std::cout << "You are really good at cuda reverse!" << std::endl;
    else
        std::cout << "Seems you cannot fully understand what cuda had done!" << std::endl;

#ifdef _MSC_VER
    system("pause");
#endif

    myExit(0);
}